
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdio>

__global__ void helloFromGPU()
{
	printf("Hello World from GPU!\n");
}

int main(void)
{
	printf("Hello World from CPU!\n");
	std::cout << "hello c++" << std::endl;
	helloFromGPU <<<1, 10>>>();
	hipDeviceReset();
	return 0;
}

