#include<stdio.h>
#include<math.h>
#include<hip/hip_runtime.h>

#define N 256

__global__ void matrix_vecter_multi_gpu_1_1(float *A_d,float *B_d,float *C_d){
  int i,j;

  for(j=0;j<N;j++){
    A_d[j]=0.0F;
    for(i=0;i<N;i++){
      A_d[j]=A_d[j]+B_d[j*N+i]*C_d[i];
    }
  }
}

int main(){
  int i,j;
  float A[N],B[N*N],C[N];
  float *A_d,*B_d,*C_d;

  dim3 blocks(1,1,1);
  dim3 threads(1,1,1);

  for(j=0;j<N;j++){
    for(i=0;i<N;i++){
      B[j*N+i]=((float)j)/256.0;
    }
  }

  for(j=0;j<N;j++){
    C[j]=1.0F;
  }

  hipMalloc((void**)&A_d,N*sizeof(float));
  hipMalloc((void**)&B_d,N*N*sizeof(float));
  hipMalloc((void**)&C_d,N*sizeof(float));

  hipMemcpy(A_d,A,N*sizeof(float),hipMemcpyHostToDevice);
  hipMemcpy(B_d,B,N*N*sizeof(float),hipMemcpyHostToDevice);
  hipMemcpy(C_d,C,N*sizeof(float),hipMemcpyHostToDevice);

  matrix_vecter_multi_gpu_1_1<<<blocks,threads>>>(A_d,B_d,C_d);

  hipMemcpy(A_d,A,N*sizeof(float),hipMemcpyDeviceToHost);

  for(j=0;j<N;j++){
    printf("A[ %d ]=%f \n",j,A[j]);
  }

  hipFree(A_d);
  hipFree(B_d);
  hipFree(C_d);


  return 0;

}
