
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<math.h>
#include<time.h>


#define N 256

void matrix_vecter_multi_cpu(float *A,float *B,float *C){
  int i,j;

  for(j=0;j<N;j++){
    A[j]=0.0F;
    for(i=0;i<N;i++){
      A[j]=A[j]+B[j*N+i]*C[i];
    }
  }
}

int main(){
  int i,j;
  float A[N],B[N*N],C[N];
  clock_t start,end;

  for(j=0;j<N;j++){
    for(i=0;i<N;i++){
      B[j*N+i]=((float)j)/256.0;
    }
  }

  for(j=0;j<N;j++){
    C[j]=1.0F;
  }

  start=clock();
  matrix_vecter_multi_cpu(A,B,C);
  end=clock();

  for(j=0;j<N;j++){
    printf("A[ %d ]=%f \n",j,A[j]);
  }

  printf("Calculation time is %lf\n",(double)(end-start)/CLOCKS_PER_SEC);

  return 0;

}
