#include<stdio.h>
#include<math.h>
#include<hip/hip_runtime.h>

#define N 256

__global__ void matrix_vector_multi_gpu_1_2(float *A_d,float *B_d,float *C_d){
  int i,j;
  int N_start;

  N_start=threadIdx.x*128;

  for(j=N_start;j<N_start+128;j++){
    A_d[j]=0.0;
    for(i=0;i<N;i++){
      A_d[j]=A_d[j]+B_d[j*N+i]*C_d[i];
    }
  }
}

int main(){
  int i,j;
  float A[N],B[N*N],C[N];
  float *A_d,*B_d,*C_d;

  dim3 blocks(1,1,1);
  dim3 threads(2,1,1);

  for(j=0;j<N;j++){
    for(i=0;i<N;i++){
      B[j*N+i]=((float)j)/256.0;
    }
  }

  for(j=0;j<N;j++){
    C[j]=1.0F;
  }

  hipMalloc((void**)&A_d,N*sizeof(float));
  hipMalloc((void**)&B_d,N*N*sizeof(float));
  hipMalloc((void**)&C_d,N*sizeof(float));

  hipMemcpy(A_d,A,N*sizeof(float),hipMemcpyHostToDevice);
  hipMemcpy(B_d,B,N*N*sizeof(float),hipMemcpyHostToDevice);
  hipMemcpy(C_d,C,N*sizeof(float),hipMemcpyHostToDevice);

  matrix_vector_multi_gpu_1_2<<<blocks,threads>>>(A_d,B_d,C_d);

  hipMemcpy(A,A_d,N*sizeof(float),hipMemcpyDeviceToHost);

  for(j=0;j<N;j++){
    printf("A[ %d ]=%f \n",j,A[j]);
  }

  hipFree(A_d);
  hipFree(B_d);
  hipFree(C_d);


  return 0;

}
