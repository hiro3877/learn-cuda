#include<stdio.h>
#include<math.h>
#include<hip/hip_runtime.h>

#define N 256

__global__ void matrix_vector_multi_gpu_1_1(float *A_d,float *B_d,float *C_d,int *tensuu_d){
  int i,j;

  printf("tensuu_d=%d\n",*tensuu_d);

  for(j=0;j<N;j++){
    A_d[j]=0.0;
    for(i=0;i<N;i++){
      A_d[j]=A_d[j]+B_d[j*N+i]*C_d[i];
    }
  }
}

int main(){
  int i,j;
  float A[N],B[N*N],C[N];
  float *A_d,*B_d,*C_d;
  int tensuu;
  tensuu=284;
  int *tensuu_d;

  hipMalloc((void**)&tensuu_d,sizeof(int));
  hipMemcpy(tensuu_d,&tensuu,sizeof(int),hipMemcpyHostToDevice);

  dim3 blocks(1,1,1);
  dim3 threads(1,1,1);

  for(j=0;j<N;j++){
    for(i=0;i<N;i++){
      B[j*N+i]=((float)j)/256.0;
    }
  }

  for(j=0;j<N;j++){
    C[j]=1.0F;
  }

  hipMalloc((void**)&A_d,N*sizeof(float));
  hipMalloc((void**)&B_d,N*N*sizeof(float));
  hipMalloc((void**)&C_d,N*sizeof(float));

  hipMemcpy(A_d,A,N*sizeof(float),hipMemcpyHostToDevice);
  hipMemcpy(B_d,B,N*N*sizeof(float),hipMemcpyHostToDevice);
  hipMemcpy(C_d,C,N*sizeof(float),hipMemcpyHostToDevice);

  matrix_vector_multi_gpu_1_1<<<blocks,threads>>>(A_d,B_d,C_d,tensuu_d);

  hipMemcpy(A,A_d,N*sizeof(float),hipMemcpyDeviceToHost);

  for(j=0;j<N;j++){
    printf("A[ %d ]=%f \n",j,A[j]);
  }

  hipFree(tensuu_d);

  hipFree(A_d);
  hipFree(B_d);
  hipFree(C_d);


  return 0;

}
