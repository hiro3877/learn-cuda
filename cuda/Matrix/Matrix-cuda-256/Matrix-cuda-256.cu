#include<stdio.h>
#include<math.h>
#include<hip/hip_runtime.h>

#define N 256

__global__ void matrix_vector_multi_gpu_1_256(float *A_d,float *B_d,float *C_d){
  int i;


  A_d[threadIdx.x]=0.0;
  for(i=0;i<N;i++){
    A_d[threadIdx.x]=A_d[threadIdx.x]+B_d[threadIdx.x*N+i]*C_d[i];
  }

}

int main(){
  int i,j;
  float A[N],B[N*N],C[N];
  float *A_d,*B_d,*C_d;

  dim3 blocks(1,1,1);
  dim3 threads(256,1,1);

  for(j=0;j<N;j++){
    for(i=0;i<N;i++){
      B[j*N+i]=((float)j)/256.0;
    }
  }

  for(j=0;j<N;j++){
    C[j]=1.0F;
  }

  hipMalloc((void**)&A_d,N*sizeof(float));
  hipMalloc((void**)&B_d,N*N*sizeof(float));
  hipMalloc((void**)&C_d,N*sizeof(float));

  hipMemcpy(A_d,A,N*sizeof(float),hipMemcpyHostToDevice);
  hipMemcpy(B_d,B,N*N*sizeof(float),hipMemcpyHostToDevice);
  hipMemcpy(C_d,C,N*sizeof(float),hipMemcpyHostToDevice);

  matrix_vector_multi_gpu_1_256<<<blocks,threads>>>(A_d,B_d,C_d);

  hipMemcpy(A,A_d,N*sizeof(float),hipMemcpyDeviceToHost);

  for(j=0;j<N;j++){
    printf("A[ %d ]=%f \n",j,A[j]);
  }

  hipFree(A_d);
  hipFree(B_d);
  hipFree(C_d);


  return 0;

}
