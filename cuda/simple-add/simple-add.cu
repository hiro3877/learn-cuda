#include <stdio.h>
#include<hip/hip_runtime.h>

__global__ void add (int a, int b, int *c) {
   *c = a+ b;
}

int main( void) {
   int c;
   int *dev_c;

   hipMalloc( (void**)&dev_c, sizeof(int) );

   add<<<1,1>>>( 12, 2000, dev_c);

   hipMemcpy( &c, dev_c, sizeof(int), hipMemcpyDeviceToHost);

   hipFree( dev_c);

   printf( "12 + 2000 = %d\n", c);

   return 0;
}
